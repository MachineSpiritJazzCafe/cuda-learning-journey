#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "utils/cuda_utils.cuh"

// Tree-based reduction with increasing stride
// VERY SLOW - operates on global memory (400-800 cycle latency)
__global__ void reduce_in_place(float* input, int n) {
    
    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Example with blockDim.x = 8:
    // Block 0: tid = 0,1,2,3,4,5,6,7  → index = 0,1,2,3,4,5,6,7
    // Block 1: tid = 0,1,2,3,4,5,6,7  → index = 8,9,10,11,12,13,14,15

    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        
        // Why? Threads read values written by OTHER threads in previous iteration
        // Without sync: Race condition - reading garbage/stale data
        __syncthreads();
        
        
        // Condition breakdown:
        //   tid % (2 * stride) == 0  → Only "even stride" threads work
        //   index + stride < n       → Boundary check (don't read out of bounds)
        if (tid % (2 * stride) == 0 && index + stride < n) {
            
            // Pattern: Each active thread sums its element with one "stride" away
            input[index] += input[index + stride];
            
            // Visual example with stride = 1 (first iteration):
            //   Thread 0: input[0] += input[1]   → input[0] = sum of [0,1]
            //   Thread 2: input[2] += input[3]   → input[2] = sum of [2,3]
            //   Thread 4: input[4] += input[5]   → input[4] = sum of [4,5]
            //   (Threads 1,3,5,7 idle - condition fails)
            
            // Visual example with stride = 2 (second iteration):
            //   Thread 0: input[0] += input[2]   → input[0] = sum of [0,1,2,3]
            //   Thread 4: input[4] += input[6]   → input[4] = sum of [4,5,6,7]
            //   (Threads 1,2,3,5,6,7 idle)
            
            // Final iteration (stride = 4):
            //   Thread 0: input[0] += input[4]   → input[0] = sum of all 8
            //   (All other threads idle)
        }
    }
    
    
    // Block results to global output:
    // Only thread 0 of each block does this
    // After the loop, input[blockIdx.x * blockDim.x] contains the block's sum
    
    if (tid == 0) {
        // Each block writes its result to the beginning of its section
        input[blockIdx.x] = input[blockIdx.x * blockDim.x];
    }
    
    // Note: After this kernel, first N_BLOCKS elements of input[] contain
    // the partial sums. Need another kernel launch or CPU code to finish.
}

// CPU Reduction Helper
float cpu_reduce(float* data, int n) {
    float sum = 0.0f;
    for (int i = 0; i < n; i++) {
        sum += data[i];
    }
    return sum;
}




// ============================================================================
// Host function: Reduce entire Array using gpu kernel only
// ============================================================================
float naiveReduceGPU(float* d_input, int n, int blockSize) {
   
     printf("\n=== Strategy: Multi-Stage GPU Reduction ===\n");
    
    int currentSize = n;
    int stage = 0;
    
    // Keep reducing until we can fit in one block
    while (currentSize > blockSize) {
        int numBlocks = (currentSize + blockSize - 1) / blockSize;
        
        printf("  Stage %d: %d elements → %d blocks\n", 
               stage, currentSize, numBlocks);
        
        reduce_in_place<<<numBlocks, blockSize>>>(d_input, currentSize);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        
        currentSize = numBlocks;
        stage++;
    }
    
    // Final reduction in one block
    printf("  Stage %d (final): %d elements → 1 block\n", stage, currentSize);
    reduce_in_place<<<1, blockSize>>>(d_input, currentSize);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Copy back final result (just 1 float)
    float result;
    CUDA_CHECK(hipMemcpy(&result, d_input, sizeof(float), 
                         hipMemcpyDeviceToHost));
    
    printf("  Total stages: %d\n", stage + 1);
    printf("  Data transfer: 4 bytes (1 float)\n");
    return result;
}

// ============================================================================
// Host function: Reduce using gpu and cpu to reduce 
// Do bulk work on GPU, finish small remainder on CPU
// Good when: Few blocks left, minimize kernel launches, CPU available
// ============================================================================
float naiveReduceHybrid(float* d_input, int n, int blockSize) {
    
    printf("\n=== Strategy: Hybrid GPU+CPU Reduction ===\n");
    
    int numBlocks = (n + blockSize - 1) / blockSize;
    
    printf("  GPU stage: %d elements → %d blocks\n", n, numBlocks);
    
    // Single GPU reduction
    reduce_in_place<<<numBlocks, blockSize>>>(d_input, n);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Copy back partial sums
    float* h_partial = new float[numBlocks];
    size_t transferBytes = numBlocks * sizeof(float);
    
    CUDA_CHECK(hipMemcpy(h_partial, d_input, transferBytes, 
                         hipMemcpyDeviceToHost));
    
    // Finish on CPU
    printf("  CPU stage: %d partial sums\n", numBlocks);
    float result = cpu_reduce(h_partial, numBlocks);
    
    printf("  Total stages: 2 (1 GPU + 1 CPU)\n");
    printf("  Data transfer: %zu bytes (%d floats)\n", 
           transferBytes, numBlocks);
    
    delete[] h_partial;
    return result;
}

// ============================================================================
// Automatically choose best strategy based on problem size
// ============================================================================
float smartReduce(float* d_input, int n, int blockSize) {
    
    // Calculate how many blocks we'd need
    int numBlocks = (n + blockSize - 1) / blockSize;
    
    // If we have more than this many blocks, use multi-stage GPU
    // otherwise, use hybrid GPU+CPU
    const int THRESHOLD = 128;
    
    float result;
    
    if (numBlocks > THRESHOLD) {
        // Large problem: Multiple GPU reductions
        printf("\nDecision: MULTI-STAGE GPU (numBlocks=%d > threshold=%d)\n", 
               numBlocks, THRESHOLD);
        printf("Reason: Many blocks → minimize data transfer, keep GPU busy\n");
        
        result = naiveReduceGPU (d_input, n, blockSize);
        
    } else {
        // Small problem: Single GPU reduction + CPU finish
        printf("\nDecision: HYBRID GPU+CPU (numBlocks=%d <= threshold=%d)\n", 
               numBlocks, THRESHOLD);
        printf("Reason: Few blocks → avoid launch overhead, CPU finish is fast\n");
        
        result = naiveReduceHybrid(d_input, n, blockSize);
    }
    
    return result;
}

int main() {
    
    printf("=============================================================\n");
    printf("Threshold-Based Reduction Strategy\n");
    printf("=============================================================\n");
    
    // Test different problem sizes to see strategy selection
    int test_sizes[] = {
        1 << 15,  // 32K   → ~128 blocks (right at threshold)
        1 << 18,  // 256K  → ~1K blocks (multi-stage)
        1 << 20,  // 1M    → ~4K blocks (multi-stage)
        1 << 22   // 4M    → ~16K blocks (multi-stage)
    };
    
    int blockSize = 256;
    
    for (int test = 0; test < 4; test++) {
        int n = test_sizes[test];
        size_t bytes = n * sizeof(float);
        
        printf("\n\n");
        printf("#############################################################\n");
        printf("TEST %d: N = %d elements (%.2f MB)\n", test + 1, n, bytes / 1e6);
        printf("#############################################################\n");
        
        // Allocate and initialize
        float* h_input = new float[n];
        for (int i = 0; i < n; i++) {
            h_input[i] = 1.0f;  // Sum should equal N
        }
        
        float* d_input;
        CUDA_CHECK(hipMalloc(&d_input, bytes));
        CUDA_CHECK(hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice));
        
        // Create CUDA events for timing
        hipEvent_t start, stop;
        CUDA_CHECK(hipEventCreate(&start));
        CUDA_CHECK(hipEventCreate(&stop));
        
        // Time the reduction
        CUDA_CHECK(hipEventRecord(start));
        float gpu_result = smartReduce(d_input, n, blockSize);
        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        
        float milliseconds = 0;
        CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
        
        // Verify
        float expected = (float)n;
        float error = fabs(gpu_result - expected) / expected;
        
        printf("\n--- Results ---\n");
        printf("GPU result:   %.0f\n", gpu_result);
        printf("Expected:     %.0f\n", expected);
        printf("Error:        %.6f%%\n", error * 100.0f);
        printf("Time:         %.4f ms\n", milliseconds);
        printf("Status:       %s\n", error < 1e-5 ? "✓ PASSED" : "✗ FAILED");
        
        // Cleanup
        CUDA_CHECK(hipEventDestroy(start));
        CUDA_CHECK(hipEventDestroy(stop));
        CUDA_CHECK(hipFree(d_input));
        delete[] h_input;
    }
    
    
    return 0;
}
